#include "hip/hip_runtime.h"
#include "Func.h"
#include <time.h>
/////////////////////////////////////
//
//  GPU Computing Project - Canny Edge Detection
//  GPU_Func.cu외 모든 Code는 수정하지 말 것
//  불가피하게 수정해야 할 경우 조교에게 문의
//
/////////////////////////////////////

int main()
{

	chrono::duration<double> c_end = chrono::duration<double>::zero();
	chrono::duration<double> g_end = chrono::duration<double>::zero();
 	chrono::system_clock::time_point time;
	int score = 0;
	double total_time = 0;

	FILE* fp = fopen("test_file.bmp", "rb");
	FILE* fp2 = fopen("gray_scale.bmp", "wb");
	FILE* fp3 = fopen("gaussian_blur.bmp", "wb");
	FILE* fp4 = fopen("sobel.bmp", "wb");
	FILE* fp5 = fopen("Non-maximum_suppression.bmp", "wb");
	FILE* fp6 = fopen("hysteresis.bmp", "wb");
	uint8_t test[200] = {0};
	fread(test, 200, 1, fp);
	
	fseek(fp, 0, SEEK_END);
	int len = ftell(fp);
	int width = 0;
	int height = 0;
	fseek(fp, 0, SEEK_SET); //go to beg.

	int * sync = NULL;
	hipMalloc(&sync,sizeof(int));
	hipFree(sync);

	uint8_t* buf = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* gray = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* gaussian = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* sobel = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* sobel_angle = (uint8_t*)malloc((len - test[10]) / 3); //malloc buffer
	uint8_t* suppression = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* hysteresis = (uint8_t*)malloc(len); //malloc buffer

	uint8_t* g_gray = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* g_gaussian = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* g_sobel = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* g_sobel_angle = (uint8_t*)malloc((len - test[10]) / 3); //malloc buffer
	uint8_t* g_suppression = (uint8_t*)malloc(len); //malloc buffer
	uint8_t* g_hysteresis = (uint8_t*)malloc(len); //malloc buffer


	memset(buf, 0, len);
	memset(gray, 0, len);
	memset(gaussian, 0, len);
	memset(sobel, 0, len);
	memset(sobel_angle, 0, (len - test[10]) / 3);
	memset(suppression, 0, len);
	memset(hysteresis, 0, len);
	memset(g_gray, 0, len);
	memset(g_gaussian, 0, len);
	memset(g_sobel, 0, len);
	memset(g_sobel_angle, 0, (len - test[10]) / 3);
	memset(g_suppression, 0, len);
	memset(g_hysteresis, 0, len);

///////////////////////////////Image Read//////////////////////////////
	fread(buf, len, 1, fp); //read into buffer
	len -= 2;
	for (int i = 0; i < test[10]; i++) {
		gray[i] = buf[i];
		gaussian[i] = buf[i];
		sobel[i] = buf[i];
		suppression[i] = buf[i];
		hysteresis[i] = buf[i];
		g_gray[i] = buf[i];
		g_gaussian[i] = buf[i];
		g_sobel[i] = buf[i];
		g_suppression[i] = buf[i];
		g_hysteresis[i] = buf[i];
	}
	for (int i = 18; i < 22; i++)
		width += test[i] * pow(256, i-18);
	for (int i = 22; i < 26; i++)
		height += test[i] * pow(256, i-22);
///////////////////////////////////////////////////////////////////////



////////////////////////////GrayScale(10)//////////////////////////////////
    time = chrono::system_clock::now();
	Grayscale(buf, gray, test[10], len);
    c_end = chrono::system_clock::now() - time;

    time = chrono::system_clock::now();
    GPU_Grayscale(buf, g_gray, test[10], len);
    g_end = chrono::system_clock::now() - time;

    printf("Gray_Scale Time\t\t\t=\tCPU(%lf)  GPU(%lf)",c_end,g_end);
    if(Image_Check(gray,g_gray,len) && c_end > g_end){
        printf("\tGray_Scale + 10(%lf)\n",g_end);
		score += 10;
		total_time += g_end.count();
    }
    else
        printf("\n");

	fwrite(gray, len+2, 1, fp2); //draw image
////////////////////////////Noise_Reduction(10)///////////////////////////////////////

    time = chrono::system_clock::now();
    Noise_Reduction(width,height,gray+test[10], gaussian+test[10]);
    c_end = chrono::system_clock::now() - time;

    time = chrono::system_clock::now();
    GPU_Noise_Reduction(width,height,gray+test[10],g_gaussian+test[10]);
    g_end = chrono::system_clock::now() - time;
 
    printf("Noise_Reduction Time\t\t=\tCPU(%lf)  GPU(%lf)",c_end,g_end);
    if(Image_Check(gaussian,g_gaussian,len) && c_end > g_end){
        printf("\tNoise_Reduction + 10(%lf)\n",g_end);
		score += 10;
		total_time += g_end.count();
    }
    else
        printf("\n");
	fwrite(gaussian, len+2, 1, fp3); //draw image

//////////////////////////Intensity_Gradient(10)////////////////////////////////////
    
    time = chrono::system_clock::now();
	Intensity_Gradient(width, height, gaussian + test[10], sobel + test[10], sobel_angle);
    c_end = chrono::system_clock::now() - time;
    
    time = chrono::system_clock::now();
    GPU_Intensity_Gradient(width,height,gaussian + test[10],g_sobel + test[10],g_sobel_angle);
    g_end = chrono::system_clock::now() - time;
	
    printf("Intensity_Gradient Time\t\t=\tCPU(%lf)  GPU(%lf)",c_end,g_end);
    if(Image_Check(sobel,g_sobel,len) && Image_Check(sobel_angle,g_sobel_angle,(len - test[10]) / 3)&& c_end > g_end){
        printf("\tIntensity_Gradient + 10(%lf)\n",g_end);
		score += 10;
		total_time += g_end.count();
    }
    else
        printf("\n");

    fwrite(sobel, len+2, 1, fp4); //draw image

//////////////////////////Non-maximum_Suppression(10)//////////////////////////////////////
	uint8_t min=255;
	uint8_t max = 0;
	uint8_t g_min = 255;
	uint8_t g_max = 0;
    
    time = chrono::system_clock::now();
	Non_maximum_Suppression(width, height, sobel_angle, sobel + test[10], suppression+test[10],min,max);
    c_end = chrono::system_clock::now() - time;
    
	
    time = chrono::system_clock::now();
    GPU_Non_maximum_Suppression(width,height,sobel_angle,sobel + test[10],g_suppression+test[10],g_min,g_max);
    g_end = chrono::system_clock::now() - time;
	
    printf("Non-maximum_Suppression Time\t=\tCPU(%lf)  GPU(%lf)",c_end,g_end);
    if(Image_Check(suppression,g_suppression,len) && c_end > g_end && g_min == min && g_max == max){
        printf("\tNon-maximum_Suppression + 10(%lf)\n",g_end);
		score += 10;
		total_time += g_end.count();
    }
    else
        printf("\n");

    fwrite(suppression, len+2, 1, fp5); //draw image

//////////////////////////Hysteresis Thresholding(10) ////////////////////////////

    time = chrono::system_clock::now();
	Hysteresis_Thresholding(width, height, suppression + test[10], hysteresis+test[10], min, max);
    c_end = chrono::system_clock::now() - time;
    
    time = chrono::system_clock::now();
    GPU_Hysteresis_Thresholding(width,height,suppression + test[10], g_hysteresis+test[10],min, max);
    g_end = chrono::system_clock::now() - time;
    
    printf("Hysteresis_Thresholding Time\t=\tCPU(%lf)  GPU(%lf)",c_end,g_end);
    if(Image_Check(hysteresis,g_hysteresis,len) && c_end > g_end){
        printf("\tHysteresis Thresholding + 10(%lf)\n",g_end);
		score += 10;
		total_time += g_end.count();
    }
    else
        printf("\n");
	
    fwrite(hysteresis, len+2, 1, fp6); //draw image
		
    printf("total_score is %d\n",score);
    if(score == 50){
      printf("execution time is %lf\n",total_time);
    }
//////////////////////////////////////////////////////////////////
	fclose(fp);
	fclose(fp2);
	fclose(fp3);
	fclose(fp4);
	fclose(fp5);
	fclose(fp6);
	free(buf);
	free(gray);
	free(gaussian);
	free(sobel);
	free(sobel_angle);
	free(suppression);
	free(hysteresis);
	free(g_gray);
	free(g_gaussian);
	free(g_sobel);
	free(g_sobel_angle);
	free(g_suppression);
	free(g_hysteresis);
	return 0;
}
