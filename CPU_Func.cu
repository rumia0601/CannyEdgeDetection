#include "hip/hip_runtime.h"
#include "Func.h"

void Grayscale(uint8_t* buf, uint8_t* gray, uint8_t start_add, int len) {
    printf("%d\n",start_add);
    for (int i = start_add; i < len; i += 3) {
		int tmp = (buf[i] * 0.114 + buf[i + 1] * 0.587 + buf[i + 2] * 0.299);
		gray[i] = tmp;
		gray[i + 1] = tmp;
		gray[i + 2] = tmp;
	}

}

float conv2d_5x5(float* filter, uint8_t* pixel, int x, int y, int width) {
	float v = 0;
	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 5; j++) {
			v += pixel[(y + i) * width + x + j] * filter[i * 5 + j];
		}
	}
	return v;
}

void Noise_Reduction(int width, int height, uint8_t *gray, uint8_t *gaussian) {
	float filter[25] = {0}; 
	float sigma = 1.0;
	for (int i = -2; i <= 2; i++) {
		for (int j = -2; j <= 2; j++) {
			filter[(i + 2) * 5 + j + 2]
				= (1 / (2 * 3.14* sigma * sigma)) * exp(-(i * i + j * j) / (2 * sigma * sigma));
		}
	}

	//zero padding
	uint8_t* tmp = (uint8_t*)malloc((width+4) * (height+4));
	memset(tmp, (uint8_t)0, (width + 4) * (height + 4));


	for (int i = 2; i < height+2; i++) {
		for (int j = 2; j < width+2; j++) {
			tmp[i * (width + 4) + j] = gray[((i - 2) * width + (j - 2)) * 3];
		}
	}
	
	//GaussianBlur

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint8_t v = conv2d_5x5(filter,tmp,j, i,width+4);
			gaussian[(i * width + j)*3] = v;
			gaussian[(i * width + j) * 3 +1] = v;
			gaussian[(i * width + j) * 3 +2] = v;
		}
	}
	free(tmp);
}
void conv2d_3x3(int* filter_y, int* filter_x, uint8_t* pixel, int x, int y, int width, int &gx, int &gy) {
	//int gx = 0;
	//int gy = 0;
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			gy += (int)pixel[(y + i) * width + x + j] * filter_y[i *3 + j];
			gx += (int)pixel[(y + i) * width + x + j] * filter_x[i * 3 + j];
		}
	}
}

void Intensity_Gradient(int width, int height, uint8_t* gaussian, uint8_t* sobel, uint8_t*angle) {
	int filter_x[9] = {-1,0,1
						,-2,0,2
						,-1,0,1};
	int filter_y[9] = {1,2,1
						,0,0,0
						,-1,-2,-1};
						
	uint8_t* tmp = (uint8_t*)malloc((width + 2) * (height + 2));
	memset(tmp, (uint8_t)0, (width + 2) * (height + 2));

	//zero padding

	for (int i = 1; i < height + 1; i++) {
		for (int j = 1; j < width + 1; j++) {
			tmp[i * (width + 2) + j] = gaussian[((i - 1) * width + (j - 1)) * 3];
		}
	}
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			int gx = 0;
			int gy = 0;
			conv2d_3x3(filter_y, filter_x, tmp, j, i, width + 2,gx,gy);
			int t = sqrt(gx * gx + gy * gy);
			uint8_t  v = 0;
			if (t > 255) {
				v = 255;
			}
			else
				v = t;

			sobel[(i * width + j) * 3] = v;
			sobel[(i * width + j) * 3 + 1] = v;
			sobel[(i * width + j) * 3 + 2] = v;
			
			float t_angle = 0;
			if(gy != 0 || gx != 0) 
				t_angle= (float)atan2(gy, gx) * 180.0 / 3.14;
			if ((t_angle > -22.5 && t_angle <= 22.5) || (t_angle > 157.5 || t_angle <= -157.5))
				angle[i * width + j] = 0;
			else if ((t_angle > 22.5 && t_angle <= 67.5) || (t_angle > -157.5 && t_angle <= -112.5))
				angle[i * width + j] = 45;
			else if ((t_angle > 67.5 && t_angle <= 112.5) || (t_angle > -112.5 && t_angle <= -67.5))
				angle[i * width + j] = 90;
			else if ((t_angle > 112.5 && t_angle <= 157.5) || (t_angle > -67.5 && t_angle <= -22.5))
				angle[i * width + j] = 135;
		}
	}
	free(tmp);
}

void Non_maximum_Suppression(int width, int height, uint8_t *angle,uint8_t *sobel, uint8_t *suppression_pixel, uint8_t& min, uint8_t& max)
{
	uint8_t p1 = 0;
	uint8_t p2 = 0;
	for (int i = 1; i < height-1; i++) {
		for (int j = 1; j < width-1; j++) {
			if (angle[i * width + j] == 0) {
				p1 = sobel[((i+1) * width + j)*3];
				p2 = sobel[((i-1) * width + j) * 3];
			}
			else if (angle[i * width + j] == 45) {
				p1 = sobel[((i + 1) * width + j-1) * 3];
				p2 = sobel[((i - 1) * width + j+1) * 3];
			}
			else if (angle[i * width + j] == 90) {
				p1 = sobel[((i) * width + j+1) * 3];
				p2 = sobel[((i) * width + j-1) * 3];
			}
			else {
				p1 = sobel[((i + 1) * width + j+1) * 3];
				p2 = sobel[((i - 1) * width + j-1) * 3];
			}
			uint8_t v = sobel[(i * width + j) * 3];
			if(min > v)
				min = v;
			if(max < v)
				max = v;
			if ((v >= p1) && (v >= p2)) {
				suppression_pixel[(i * width + j) * 3] = v;
				suppression_pixel[(i * width + j) * 3 + 1] = v;
				suppression_pixel[(i * width + j) * 3 + 2] = v;
			}
			else {
				suppression_pixel[(i * width + j) * 3] = 0;
				suppression_pixel[(i * width + j) * 3 + 1] = 0;
				suppression_pixel[(i * width + j) * 3 + 2] = 0;
			}
		}
	}
}

void Hysteresis_check(int width, int height, int x, int y, uint8_t * hysteresis, uint8_t *tmp_hysteresis){
	for (int i = y-1; i < y+2; i++) {
		for (int j = x-1; j < x+2; j++) {
			if ((i < height && j < width) && (i >= 0 && j >= 0)) {
				if (tmp_hysteresis[(i * width + j)*3] == 255) {
					hysteresis[(y * width + x)*3] = 255;
					hysteresis[(y * width + x) * 3+1] = 255;
					hysteresis[(y * width + x) * 3+2] = 255;
					return;
				}
			}
		}
	}
}

void Hysteresis_Thresholding(int width, int height, uint8_t *suppression_pixel,uint8_t *hysteresis, uint8_t min, uint8_t max) {
	uint8_t diff = max - min;
	uint8_t low_t = min + diff * 0.01;
	uint8_t high_t = min + diff * 0.2;
	uint8_t *tmp_hysteresis = (uint8_t*)malloc(sizeof(uint8_t)*width*height*3);

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint8_t v = suppression_pixel[(i * width + j)*3];
			if (v < low_t) {
				hysteresis[(i * width + j) * 3] = 0;
				hysteresis[(i * width + j) * 3+1] = 0;
				hysteresis[(i * width + j) * 3+2] = 0;
			}
			else if (v < high_t) {
				hysteresis[(i * width + j) * 3] = 123;
				hysteresis[(i * width + j) * 3 + 1] = 123;
				hysteresis[(i * width + j) * 3 + 2] = 123;
			}
			else {
				hysteresis[(i * width + j) * 3] = 255;
				hysteresis[(i * width + j) * 3 + 1] = 255;
				hysteresis[(i * width + j) * 3 + 2] = 255;
			}
		}
	}
//////////////////////Modified in Version3//////////////////////////////
	memcpy(tmp_hysteresis,hysteresis,sizeof(uint8_t)*width*height*3);
    	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if(tmp_hysteresis[(i*width+j)*3] == 123){
                		Hysteresis_check(width,height,j,i,hysteresis,tmp_hysteresis);
			}
		}
	}
//////////////////////////////////////////////////////////////////////
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if (hysteresis[(i * width + j) * 3] != 255) {
				hysteresis[(i * width + j) * 3] = 0;
				hysteresis[(i * width + j) * 3+1] = 0;
				hysteresis[(i * width + j) * 3+2] = 0;
			}
		}
	}
	free(tmp_hysteresis);
}


bool Image_Check(uint8_t *cpu, uint8_t *gpu, int len){
    for(int i=0;i<len;i++){
        if(cpu[i] != gpu[i] && cpu[i] + 1 != gpu[i] && cpu[i] - 1 != gpu[i]){ 
            return false;
        }
    }
    return true;
}
